#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_helpers.h"
#define N 4096

__global__ void vector_add_kernel(int *d_a, int *d_b, int *d_c) {
    int block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_idx = block_idx * blockDim.y * blockDim.x + threadIdx.x + threadIdx.y * blockDim.x;

    if (thread_idx < N) {

        printf("Hello block %i (x %i, y %i, z %i) running thread %i (x %i, y %i, z %i)\n",
                block_idx, blockIdx.x, blockIdx.y, blockIdx.z,
                thread_idx, threadIdx.x, threadIdx.y, threadIdx.z);

        d_c[thread_idx] = d_a[thread_idx] + d_b[thread_idx];
    }
}

int main() {
    struct hipDeviceProp_t props;
    int status = hipGetDeviceProperties(&props, 0);

    printf("Threads per block: (%i %i %i)\n",
            props.maxThreadsDim[0],
            props.maxThreadsDim[1],
            props.maxThreadsDim[2]);

    dim3 grid(4, 8);
    dim3 block(128);

    int h_a[N];
    int h_b[N];
    int h_c[N];
    int *d_a, *d_b, *d_c;

    for (int i=0; i<N; i++) {
        h_a[i] = i;
        h_b[i] = i + 1;
    }

    CUDA_CALL(hipMalloc(&d_a, N * sizeof(int)));
    CUDA_CALL(hipMalloc(&d_b, N * sizeof(int)));
    CUDA_CALL(hipMalloc(&d_c, N * sizeof(int)));

    CUDA_CALL(hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice));

    hipEvent_t start;
    hipEvent_t stop;
    float elapsedTime;

    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    CUDA_CALL(hipEventRecord(start, 0));

    vector_add_kernel<<<grid, block>>>(d_a, d_b, d_c);
    cuda_peek();

    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&elapsedTime, start, stop));

    CUDA_CALL(hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost));

    for (int i=0; i<N; i++) {
        printf("%i+%i = %i\n", h_a[i], h_b[i], h_c[i]);
    }

    printf("Time taken: %f ms\n", elapsedTime);

    CUDA_CALL(hipFree(d_a));
    CUDA_CALL(hipFree(d_b));
    CUDA_CALL(hipFree(d_c));

    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

    CUDA_CALL(hipDeviceReset());
    return 0;
}
