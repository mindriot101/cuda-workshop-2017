#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 64

__global__ void matrix_add_kernel(int *d_a, int *d_b, int *d_c) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int index = row * N + col;

    if ((row < N) && (col < N)) {
        int aval = d_a[index];
        int bval = d_b[index];
        int result = aval + bval;
        /* printf("row %i col %i a: %d, b: %d, result: %d\n", row, col, aval, bval, result); */
        d_c[index] = result;
    }
}

__global__ void matrix_mult_kernel(int *d_a, int *d_b, int *d_c) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    /* int index = row * N + col; */


    if ((row < N) && (col < N)) {
        int result = 0;
        for (int k=0; k<N; k++) {
            result += d_a[row * N + k] * d_b[k * N + col];
        }
        d_c[row * N + col] = result;
    }
}

void matrix_mult(int *a, int *b, int *c) {
    int row, col, k, sum;
    for (row=0; row<N; row++) {
        for (col=0; col<N; col++) {
            sum = 0;
            for (k=0; k<N; k++) {
                sum += a[row * N + k] * b[k * N + col];
            }
            c[row * N + col] = sum;
        }
    }
}

void matrix_add(int *a, int *b, int *c) {
    int index;
    for (int col=0; col<N; col++) {
        for (int row=0; row<N; row++) {
            index = row * N + col;
            c[index] = a[index] + b[index];
        }
    }
}

void print_matrix(int *matrix, const char *name) {
    printf("Matrix %s\n", name);
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            printf("(%i,%i) = %i\n", j, i, matrix[i * N + j]);
        }
    }
}

int main() {
    int *h_a, *h_b, *h_c, *h_d;
    int *d_a, *d_b, *d_c;

    int size = N * N * sizeof(int);
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);
    h_d = (int*)malloc(size);

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            h_a[i * N + j] = i + 1;
            h_b[i * N + j] = i + 1;
            h_c[i * N + j] = 0;
        }
    }

    /* print_matrix(h_a, "Host input A"); */
    /* print_matrix(h_b, "Host input B"); */

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    float deviceElapsedTime, hostElapsedtime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dim3 grid(4, 4);
    dim3 block(32, 32);
    matrix_mult_kernel<<<grid, block>>>(d_a, d_b, d_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&deviceElapsedTime, start, stop);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    printf("Time taken: %f ms\n", deviceElapsedTime);

    hipEventRecord(start, 0);

    matrix_mult(h_a, h_b, h_d);

    /* print_matrix(h_d, "Host result"); */
    /* print_matrix(h_c, "Device result"); */

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&hostElapsedtime, start, stop);

    printf("Time taken: %f ms\n", hostElapsedtime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i=0; i<N*N; i++) {
        if (h_c[i] != h_d[i]) {
            printf("Error: CPU and GPU results do not match at index %d, %d != %d\n",
                    i, h_c[i], h_d[i]);
            exit(1);
        }
    }

    printf("Speedup: %f\n", hostElapsedtime / deviceElapsedTime);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceReset();
    return 0;
}
